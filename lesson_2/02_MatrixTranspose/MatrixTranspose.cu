#include "hip/hip_runtime.h"
#include <chrono>
#include <iomanip>
#include <iostream>
#include <random>
#include "Timer.cuh"
#include "CheckError.cuh"
using namespace timer;

const int TILE_WIDTH = 16;
const int BLOCK_SIZE = TILE_WIDTH;

__global__
void matrixTransposeKernel(const int* d_matrix_in,
                           int        N,
                           int*       d_matrix_out) {
	__shared__ int ds_M[TILE_WIDTH][TILE_WIDTH];

	// row and column of product matrix element to work on
	int Row = blockIdx.y * blockDim.y + threadIdx.y;
	int Col = blockIdx.x * blockDim.x + threadIdx.x;
	if (Col < N && Row < N) {

		ds_M[threadIdx.y][threadIdx.x] = d_matrix_in[Row * N + Col];

		__syncthreads();

		Col = blockIdx.y * blockDim.y + threadIdx.x;
		Row = blockIdx.x * blockDim.y + threadIdx.y;

		d_matrix_out[Row*N+Col] = ds_M[threadIdx.x][threadIdx.y]; 
		
		// d_matrix_out[Col * N + Row] = ds_M[threadIdx.y][threadIdx.x];
	}
}

const int N  = 8192;

int main() {
    Timer<DEVICE> TM_device;
    Timer<HOST>   TM_host;
    // -------------------------------------------------------------------------
    // HOST MEMORY ALLOCATION
    int* h_matrix_in  = new int[N * N];
    int* h_matrix_tmp = new int[N * N]; // <-- used for device result
    int* h_matrix_out = new int[N * N];

    // -------------------------------------------------------------------------
    // HOST INITILIZATION
    unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
    std::default_random_engine generator(seed);
    std::uniform_int_distribution<int> distribution(1, 100);

    for (int i = 0; i < N * N; i++)
        h_matrix_in[i] = distribution(generator);

    // -------------------------------------------------------------------------
    // HOST EXECUTIION
    TM_host.start();

    for (int i = 0; i < N ; i++) {
        for (int j = 0; j < N ; j++)
            h_matrix_out[i * N + j] = h_matrix_in[j * N + i];
    }

    TM_host.stop();
    TM_host.print("MatrixTranspose host:   ");

    // -------------------------------------------------------------------------
    // DEVICE MEMORY ALLOCATION
    int *d_matrix_in, *d_matrix_out;
    SAFE_CALL( hipMalloc( &d_matrix_in, N * N * sizeof(int)) )
    SAFE_CALL( hipMalloc( &d_matrix_out, N * N * sizeof(int)) )

    // -------------------------------------------------------------------------
    // COPY DATA FROM HOST TO DEVIE
    SAFE_CALL( hipMemcpy( d_matrix_in, h_matrix_in, N * N * sizeof(int), hipMemcpyHostToDevice ) )

    // -------------------------------------------------------------------------
    // DEVICE EXECUTION
    dim3 block_size( BLOCK_SIZE, BLOCK_SIZE, 1 );
    dim3 num_blocks( N/BLOCK_SIZE, N/BLOCK_SIZE, 1 );

    TM_device.start();
	
    matrixTransposeKernel<<< num_blocks, block_size>>>(d_matrix_in, N, d_matrix_out);

    TM_device.stop();
    CHECK_CUDA_ERROR
    TM_device.print("MatrixTranspose device: ");

    std::cout << std::setprecision(1)
              << "Speedup: " << TM_host.duration() / TM_device.duration()
              << "x\n\n";

    // -------------------------------------------------------------------------
    // COPY DATA FROM DEVICE TO HOST
    SAFE_CALL( hipMemcpy( h_matrix_tmp, d_matrix_out, N * N * sizeof(int), hipMemcpyDeviceToHost ) )

    // -------------------------------------------------------------------------
    // RESULT CHECK
    /*
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            std::cout << h_matrix_in[i * N + j] << "\t";
        }
	std::cout << std::endl;
    }

    std::cout << "\n\n\n" << std::endl;

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            std::cout << h_matrix_tmp[i * N + j] << "\t";
        }
	std::cout << std::endl;
    }

    std::cout << "\n\n\n" << std::endl;

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            std::cout << h_matrix_out[i * N + j] << "\t";
        }
	std::cout << std::endl;
    }
    */

    for (int i = 0; i < N * N; i++) {
        if (h_matrix_out[i] != h_matrix_tmp[i]) {
            std::cerr << "wrong result at: ("
                      << (i / N) << ", " << (i % N) << ")"
                      << "\nhost:   " << h_matrix_out[i]
                      << "\ndevice: " << h_matrix_tmp[i] << "\n\n";
            hipDeviceReset();
            std::exit(EXIT_FAILURE);
        }
    }
    std::cout << "<> Correct\n\n";

    // -------------------------------------------------------------------------
    // HOST MEMORY DEALLOCATION
    delete[] h_matrix_in;
    delete[] h_matrix_out;
    delete[] h_matrix_tmp;

    // -------------------------------------------------------------------------
    // DEVICE MEMORY DEALLOCATION
    SAFE_CALL( hipFree( d_matrix_in ) )
    SAFE_CALL( hipFree( d_matrix_out ) )

    // -------------------------------------------------------------------------
    hipDeviceReset();
}
