#include "hip/hip_runtime.h"
#include <chrono>
#include <iomanip>
#include <iostream>
#include <random>
#include "Timer.cuh"
#include "CheckError.cuh"
using namespace timer;

const int BLOCK_SIZE_X = 16;
const int BLOCK_SIZE_Y = 16;

__global__
void matrixMultiplicationKernel(const int* d_matrixA,
                                const int* d_matrixB,
                                int        N,
                                int*       d_matrixC) {
	int Row = blockIdx.y*blockDim.y + threadIdx.y;
	int Col = blockIdx.x*blockDim.x + threadIdx.x;
	int tmpval = 0;
	for (int k = 0; k < N; ++k) {
		tmpval += d_matrixA[Row*N+k] + d_matrixB[Col+k*N];	
	}
	d_matrixC[Row*N+Col] = tmpval;
}

const int N = 8;

int main() {
    Timer<DEVICE> TM_device;
    Timer<HOST>   TM_host;
    // -------------------------------------------------------------------------
    // HOST MEMORY ALLOCATION
    int* h_matrixA    = new int[N * N];
    int* h_matrixB    = new int[N * N];
    int* h_matrix_tmp = new int[N * N]; // <-- used for device result
    int* h_matrixC    = new int[N * N];

    // -------------------------------------------------------------------------
    // HOST INITILIZATION
    unsigned seed = std::chrono::system_clock::now().time_since_epoch().count();
    std::default_random_engine generator(seed);
    std::uniform_int_distribution<int> distribution(1, 100);

    for (int i = 0; i < N * N; i++) {
        h_matrixA[i] = distribution(generator);
        h_matrixB[i] = distribution(generator);
    }
    // -------------------------------------------------------------------------
    // HOST EXECUTIION
    TM_host.start();

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            int sum = 0;
            for (int k = 0; k < N; k++)
                 sum += h_matrixA[i * N + k] * h_matrixB[k * N + j];
            h_matrixC[i * N + j] = sum;
        }
    }

    TM_host.stop();
    TM_host.print("MatrixMultiplication host:   ");

    // -------------------------------------------------------------------------
    // DEVICE MEMORY ALLOCATION
    int *d_matrixA, *d_matrixB, *d_matrixC;
    SAFE_CALL( hipMalloc( &d_matrixA, N * N * sizeof(int) ) )
    SAFE_CALL( hipMalloc( &d_matrixB, N * N * sizeof(int) ) )
    SAFE_CALL( hipMalloc( &d_matrixC, N * N * sizeof(int) ) )

    // -------------------------------------------------------------------------
    // COPY DATA FROM HOST TO DEVIE
    SAFE_CALL( hipMemcpy( d_matrixA, h_matrixA, N * N * sizeof(int), hipMemcpyHostToDevice ) )
    SAFE_CALL( hipMemcpy( d_matrixB, h_matrixB, N * N * sizeof(int), hipMemcpyHostToDevice ) )

    // -------------------------------------------------------------------------
    // DEVICE EXECUTION
    int tilew = 8;

    TM_device.start();

    dim3 block_size( N/tilew, N/tilew, 1 );
    dim3 num_blocks( tilew, tilew, 1 );
    matrixMultiplicationKernel<<< num_blocks, block_size >>>(d_matrixA, d_matrixB, N, d_matrixC);

    TM_device.stop();
    CHECK_CUDA_ERROR
    TM_device.print("MatrixMultiplication device: ");

    std::cout << std::setprecision(1)
              << "Speedup: " << TM_host.duration() / TM_device.duration()
              << "x\n\n";

    // -------------------------------------------------------------------------
    // COPY DATA FROM DEVICE TO HOST
    SAFE_CALL( hipMemcpy(h_matrix_tmp, d_matrixC, N * N * sizeof(int), hipMemcpyDeviceToHost) )

    // -------------------------------------------------------------------------
    // RESULT CHECK
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
        	std::cout << h_matrixA[i*N+j] << "\t";
	}   
	std::cout << std::endl;
    }
    std::cout << std::endl;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
        	std::cout << h_matrixB[i*N+j] << "\t";
	}   
	std::cout << std::endl;
    }
    std::cout << std::endl;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
        	std::cout << h_matrixC[i*N+j] << "\t";
	}   
	std::cout << std::endl;
    }
    std::cout << std::endl;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
        	std::cout << h_matrix_tmp[i*N+j] << "\t";
	}   
	std::cout << std::endl;
    }
    std::cout << std::endl;


    for (int i = 0; i < N * N; i++) {
        if (h_matrixC[i] != h_matrix_tmp[i]) {
            std::cerr << "wrong result at: ("
                      << (i / N) << ", " << (i % N) << ")"
                      << "\nhost:   " << h_matrixC[i]
                      << "\ndevice: " << h_matrix_tmp[i] << "\n\n";
            hipDeviceReset();
            std::exit(EXIT_FAILURE);
        }
    }

    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
        	std::cout << h_matrixC[i*N+j] << "\t";
	}   
	std::cout << std::endl;
    }

    std::cout << "<> Correct\n\n";

    // -------------------------------------------------------------------------
    // HOST MEMORY DEALLOCATION
    delete[] h_matrixA;
    delete[] h_matrixB;
    delete[] h_matrixC;
    delete[] h_matrix_tmp;

    // -------------------------------------------------------------------------
    // DEVICE MEMORY DEALLOCATION
    SAFE_CALL( hipFree( d_matrixA ) )
    SAFE_CALL( hipFree( d_matrixB ) )
    SAFE_CALL( hipFree( d_matrixC ) )

    // -------------------------------------------------------------------------
    hipDeviceReset();
}
